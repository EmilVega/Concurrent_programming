#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
//#include <graphics.h>
#include <math.h>

#include <iostream>
#include "loadImage.h"
#include <ctime>
//#include "PRINT_MATRIX.h"
int start_s = clock();
const int st = 2;
char dummy[1];
double M[101][101];
double mResult[50][50];

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecMult(double *a, double *b, double *c, int n)
{
	// Get our global thread ID
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	// Make sure we do not go out of bounds
	if (id < n)
		c[id] = a[id] * b[id];
}


int main(int argc, char* argv[])
{
	
	cargarImagenes();
	
	const int fl = (sizeof matImg / sizeof matImg[0]) + 1;
	const int cl = (sizeof matImg[0] / sizeof(int)) + 1;

	
	
	// Size of vectors
	int n = 9;

	//Pass to matrux with 0s at the end
	for (int i = 0; i < fl; i++) {
		for (int j = 0; j < cl; j++) {
			if (j == cl - 1 || i == fl - 1) { M[i][j] = 0; }
			else { M[i][j] = matImg[i][j]; }
		}
	}

	/*std::cout << "\n Matriz origen is: \n";
	for (int i = 0; i<fl; i++) {
		for (int j = 0; j<cl; j++) {
			std::cout << M[i][j] << " ";
		}
		std::cout << "\n";
	}*/

	

	// Host input vectors
	double *f1;
	double *f2;
	double *f3;
	double *f4;
	double *sbm;

	//double *h_a;
	//double *h_b;

	//Host output vector
	double *r_c;

	// Device input vectors
	double *d_f1;
	double *d_f2;
	double *d_f3;
	double *d_f4;
	double *d_sbm;

	//Device output vector
	double *d_rc;

	// Size, in bytes, of each vector
	size_t bytes = n * sizeof(double);

	// Allocate memory for each vector on host
	f1 = (double*)malloc(bytes);
	f2 = (double*)malloc(bytes);
	f3 = (double*)malloc(bytes);
	f4 = (double*)malloc(bytes);
	sbm = (double*)malloc(bytes);
	r_c = (double*)malloc(bytes);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_f1, bytes);
	hipMalloc(&d_f2, bytes);
	hipMalloc(&d_f3, bytes);
	hipMalloc(&d_f4, bytes);
	hipMalloc(&d_sbm, bytes);
	hipMalloc(&d_rc, bytes);

	//int i;
	// Initialize vectors on host

	f1[0] = -1; f1[1] = 0; f1[2] = 1; f1[3] = -1; f1[4] = 0; f1[5] = 1; f1[6] = -1; f1[7] = 0; f1[8] = 1;
	f2[0] = -1; f2[1] = -1; f2[2] = -1; f2[3] = 0; f2[4] = 0; f2[5] = 0; f2[6] = 1; f2[7] = 1; f2[8] = 1;
	f3[0] = 0; f3[1] = 1; f3[2] = 1; f3[3] = -1; f3[4] = 0; f3[5] = 1; f3[6] = -1; f3[7] = -1; f3[8] = 0;
	f4[0] = 1; f4[1] = 1; f4[2] = 0; f4[3] = 1; f4[4] = 0; f4[5] = -1; f4[6] = 0; f4[7] = -1; f4[8] = -1;

	/*for (i = 0; i < n; i++) {
	h_a[i] = sin(i)*sin(i);
	h_b[i] = cos(i)*cos(i);
	}*/


	// Copy host vectors to device
	hipMemcpy(d_f1, f1, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_f2, f2, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_f3, f4, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_f4, f4, bytes, hipMemcpyHostToDevice);

	int blockSize, gridSize;

	// Number of threads in each thread block
	blockSize = 1024;

	// Number of thread blocks in grid
	gridSize = (int)ceil((float)n / blockSize);

	// Execute the kernel
	
	int f = 0, c=0;
	for (int i = 1; i < fl - 1; i += st) {
		c = 0;
		for (int j = 1; j < cl - 1; j += st) {
			sbm[0] = M[i - 1][j - 1];	sbm[1] = M[i - 1][j];	sbm[2] = M[i - 1][j + 1];
			sbm[3] = M[i][j - 1];		sbm[4] = M[i][j];		sbm[5] = M[i][j + 1];
			sbm[6] = M[i + 1][j - 1];	sbm[7] = M[i + 1][j];	sbm[8] = M[i + 1][j + 1];

			hipMemcpy(d_sbm, sbm, bytes, hipMemcpyHostToDevice);

			vecMult << <gridSize, blockSize >> > (d_sbm, d_f1, d_rc, n);

			// Copy array result back to host
			hipMemcpy(r_c, d_rc, bytes, hipMemcpyDeviceToHost);

			double resultado1 = 0;
			for (int k = 0; k < n; k++) {
				resultado1 += r_c[k];
			}
			
			vecMult << <gridSize, blockSize >> > (d_sbm, d_f2, d_rc, n);

			
			// Copy array result back to host
			hipMemcpy(r_c, d_rc, bytes, hipMemcpyDeviceToHost);
          
			
			double resultado2 = 0;
			for (int k = 0; k < n; k++) {
				resultado2 += r_c[k];
			}

			
			vecMult << <gridSize, blockSize >> > (d_sbm, d_f3, d_rc, n);

			// Copy array result back to host
			hipMemcpy(r_c, d_rc, bytes, hipMemcpyDeviceToHost);
			
			double resultado3 = 0;
			for (int k = 0; k < n; k++) {
				resultado3 += r_c[k];
			}

			
			vecMult << <gridSize, blockSize >> > (d_sbm, d_f4, d_rc, n);

			// Copy array result back to host
			hipMemcpy(r_c, d_rc, bytes, hipMemcpyDeviceToHost);
			
			double resultado4 = 0;
			for (int k = 0; k < n; k++) {
				resultado4 += r_c[k];
			}

			
			mResult[f][c] = resultado1 + resultado2 + resultado3 + resultado4;

			

			c++;
			
		}
		f++;
		//getchar();
	}


	/*std::cout << "\n Matriz resultante is: \n";
	for (int i = 0; i<fl / 2; i++) {
		for (int j = 0; j<cl / 2; j++) {
			std::cout << mResult[i][j] << " ";
		}
		std::cout << "\n";
	}*/



	FILE* fichero;
	fichero = fopen("mat.dat", "wt");
	for (int i = 0; i<fl / 2; i++) {
		for (int j = 0; j<cl / 2; j++) {
			fprintf(fichero,"%f\n", mResult[i][j]);
		}
	}
	fclose(fichero);
	/*
		fprintf(fichero, "%s%s%sEscritas %d+1 lineas en fichero", cadena1, cadena2, cadena3, i);
	fclose(fichero);
	fputs("Aprender a programar (linea 1)\n", fichero);
	fputs("requiere esfuerzo (linea 2)\n", fichero);
	fputs("y dedicacion (linea 3)", fichero);
	fclose(fichero);
	printf("Proceso completado");*/

	// Release device memory
	hipFree(d_f1);
	hipFree(d_f2);
	hipFree(d_f3);
	hipFree(d_f4);
	hipFree(d_sbm);
	hipFree(d_rc);

	// Release host memory
	free(f1);
	free(f2);
	free(f3);
	free(f4);
	free(sbm);
	free(r_c);

	// -------------------- del main de print matrix

	
	/*train();
	clrscr();*/


	int stop_s = clock();
	std::cout << "el tiempo de este c�digo fue: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000;


	getch();

	return 0;
}